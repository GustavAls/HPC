#include "hip/hip_runtime.h"
#define STRIDE 4
extern "C"{
    __global__ void kernel4(double *A, double *B, double *C,int k,int m,int n){
        int i, j, q, l;
        double sum;
            
        //Computing the global coordinates of the thread
        int i = (blockIdx.y * blockDim.y + threadIdx.y) * STRIDE;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < m && j < n)
        {
            for(q = 0; q < k; q++){
                sum = 0.0;
                for(l = 0; l < STRIDE; l++){
                    if (i + l < m)
                        sum += A[(i+l)*k + q] * B[q*n + j];
                }
                C[(i+l)*n+j] = sum;
            }
        }
    }
}

extern "C" {
    void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;
        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        int d1,d2;
        int block_size = 16;
                
        //Assigning a 2D thread grid in each block
        dim3 threadsPerBlock(block_size, block_size);

        //Defining the grid layout depending on the input dimensions
        d1 = (int) (m - 1) / block_size + 1;
        d2 = (int) (n / STRIDE - 1) / block_size + 1;

        //Defining grid size
        dim3 blocksPerGrid(d1, d2);

        kernel3_below<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        //kernel3_right<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        
        hipDeviceSynchronize();

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
      }
  }