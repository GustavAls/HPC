
#include <hip/hip_runtime.h>
/*
Implementation delegating x elements of C to each thread
*/

//Version with 4 elements pr thread
__global__ void kernel4_4(int m,int n, int k, double *A, double *B, double *C){
    int i, j, q;
    double sum1 = 0.0, sum2 = 0.0, sum3 = 0.0, sum4 = 0.0;
        
    //Computing the global coordinates of the thread
    i = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < m && j < n)
    {
        for(q = 0; q < k; q++){
            sum1 += A[i*k + q] * B[q*n + j];
            if (i+1 < m) sum2 += A[(i+1)*k + q] * B[q*n + j];
            if (i+2 < m) sum3 += A[(i+2)*k + q] * B[q*n + j];
            if (i+3 < m) sum4 += A[(i+3)*k + q] * B[q*n + j];
        }
        C[i*n+j] = sum1;
        if (i+1 < m) C[(i+1)*n+j] = sum2;
        if (i+2 < m) C[(i+2)*n+j] = sum3;
        if (i+3 < m) C[(i+3)*n+j] = sum4;
    }
}

//Version with 6 elements pr thread
__global__ void kernel4_6(int m,int n, int k, double *A, double *B, double *C){
    int i, j, q;
    double sum1 = 0.0, sum2 = 0.0, sum3 = 0.0, sum4 = 0.0, sum5 = 0.0, sum6 = 0.0;
        
    //Computing the global coordinates of the thread
    i = 6 * (blockIdx.x * blockDim.x + threadIdx.x);
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < m && j < n)
    {
        for(q = 0; q < k; q++){
            sum1 += A[i*k + q] * B[q*n + j];
            if (i+1 < m) sum2 += A[(i+1)*k + q] * B[q*n + j];
            if (i+2 < m) sum3 += A[(i+2)*k + q] * B[q*n + j];
            if (i+3 < m) sum4 += A[(i+3)*k + q] * B[q*n + j];
            if (i+4 < m) sum5 += A[(i+4)*k + q] * B[q*n + j];
            if (i+5 < m) sum6 += A[(i+5)*k + q] * B[q*n + j];
        }
        C[i*n+j] = sum1;
        if (i+1 < m) C[(i+1)*n+j] = sum2;
        if (i+2 < m) C[(i+2)*n+j] = sum3;
        if (i+3 < m) C[(i+3)*n+j] = sum4;
        if (i+4 < m) C[(i+4)*n+j] = sum5;
        if (i+5 < m) C[(i+5)*n+j] = sum6;
    }
}

extern "C" {
    void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;
        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        int d1,d2;
        int block_size = 16;
        int num_elements = 4;
                
        //Assigning a 2D thread grid in each block
        dim3 threadsPerBlock(block_size, block_size);

        //Defining the grid layout depending on the input dimensions
 
        d1 = (int) (m / num_elements - 1) / block_size + 1;
        d2 = (int) (n - 1) / block_size + 1;

        //Defining grid size
        dim3 blocksPerGrid(d1, d2);

        kernel4_4<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        //kernel4_6<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        
        hipDeviceSynchronize();

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
      }
  }