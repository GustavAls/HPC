#include "hip/hip_runtime.h"
/*
Implementation letting each thread compute exatcly two elements of C
*/

//Version 1: right neighbor
__global__ void kernel3_right(int m, int n, int k, double *A, double *B, double *C) {
    int i, j, q;
    double sum1 = 0.0,sum2 = 0.0;

    //Computing the global coordinates of the thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = 2 * (blockIdx.y * blockDim.y + threadIdx.y);
    
    if (i < m && j < n){
        for (q = 0; q < k; q++) {
            //Compute product for first column in B
            sum1 += A[i*k + q] * B[q*n + j];
            if (j+1 < n) 
                //Compute product for second column in B
                sum2 += A[i*k + q] * B[q*n + j+1];
        }

    C[i*n + j] = sum1;
    if (j+1 < n) 
        C[i*n + j + 1] = sum2;
    }
}


//Version 2: below neighbor
__global__ void kernel3_below(int m, int n, int k, double *A, double *B, double *C) {

    int i, j, q;
    double sum1 = 0.0,sum2 = 0.0;

    //Computing the global coordinates of the thread
    i = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < m && j < n){
        for (q = 0; q < k; q++) {
            sum1 += A[i*k + q] * B[q*n + j];
            if (i+1 < m) 
                sum2 += A[(i+1)*k + q] * B[q*n + j];
        }
    
    C[i*n + j] = sum1;
    if (i+1 < m) 
        C[(i+1)*n + j] = sum2;
    }
}



extern "C" {
    void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;
        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        int d1,d2;
        int block_size = 16;
                
        //Assigning a 2D thread grid in each block
        dim3 threadsPerBlock(block_size, block_size);

        //Defining the grid layout depending on the input dimensions
        //For right
        // d1 = (int) (m - 1) / block_size + 1;
        // d2 = (int) (n / 2 - 1) / block_size + 1;
        
        //For below
        d1 = (int) (m / 2 - 1) / block_size + 1;
        d2 = (int) (n - 1) / block_size + 1;


        //Defining grid size
        dim3 blocksPerGrid(d1, d2);
        
        double start = omp_get_wtime();
        //kernel3_right<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        kernel3_below<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        double seconds = omp_get_wtime() - start;
		printf("Run time (s): %f", seconds);
        
        hipDeviceSynchronize();

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
      }
  }