
#include <hip/hip_runtime.h>
/*
Implementation of matrix multiplication which utilizes shared memory
*/

__global__ void kernel5(int m, int n, int k, double *A, double *B, double *C) {
    
    //Utilizing dynamic shared memory allocation
    extern __shared__ double blocks[];

    __shared__ double* Asub;
    Asub =  &blocks[0];
    __shared__ double* Bsub;
    Bsub =  &blocks[blockDim.x*blockDim.y];

    int q, j;
    double sum;
    const int side = blockDim.x;

    int A_toprow = blockIdx.y*blockDim.y*k;
    int B_topcol = blockIdx.x*blockDim.x;

    for (q = 0; q < k; q += side) {

        int A_toprow_block = A_toprow + q;
        int B_topcol_block = B_topcol + q*n;

        Asub[threadIdx.y*side + threadIdx.x] = A[A_toprow_block + threadIdx.y*k + threadIdx.x];
        Bsub[threadIdx.y*side + threadIdx.x] = B[B_topcol_block + threadIdx.y*n + threadIdx.x];

        __syncthreads();

        sum = 0.0;
        for (j = 0; j < side; j++) {
            sum += Asub[threadIdx.y*side + j] * Bsub[side*j + threadIdx.x];
        }

        //Barrier to ensure not race conditions are present for accessing Asub and Bsub
        __syncthreads();

       C[blockIdx.y*blockDim.y*n + threadIdx.y*n + blockIdx.x*blockDim.x + threadIdx.x] += sum;

    }
}

extern "C" {
    void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;
        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        int d1,d2;
        int block_size = 16;
                
        //Assigning a 2D thread grid in each block
        dim3 threadsPerBlock(block_size, block_size);

        //Defining the grid layout depending on the input dimensions
        d1 = (int) (m - 1) / block_size + 1;
        d2 = (int) (n - 1) / block_size + 1;

        //Defining grid size
        dim3 blocksPerGrid(d1, d2);

        kernel5<<<blocksPerGrid,threadsPerBlock, (2* threadsPerBlock.x * threadsPerBlock.y * sizeof(double))>>>(m, n, k, A_d, B_d, C_d);
        
        hipDeviceSynchronize();

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
    }
}