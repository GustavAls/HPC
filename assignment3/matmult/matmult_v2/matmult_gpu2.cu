
#include <hip/hip_runtime.h>
/*
Implementation using one thread pr. element in C
*/
__global__ void kernel2(int m, int n, int k, double *A, double *B, double *C) {
    double sum = 0;
    int i, j, q;

    //Computing the global coordinates of the thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
  
    //Checking for out of bounds
    if (i < m && j < n){
        //Multiplitying a row in A with a column in B, corresponding to one element in C
        for (q = 0; q < k; q++) {
            sum += A[i*k + q] * B[q*n + j];
        }
        C[i*n + j] = sum;
    }
}
  
  
extern "C" {
    void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;
        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        int d1,d2;
        int block_size = 16;
                
        //Assigning a 2D thread grid in each block
        dim3 threadsPerBlock(block_size, block_size);

        //Defining the grid layout depending on the input dimensions
        d1 = (int) (m - 1) / block_size + 1;
        d2 = (int) (n - 1) / block_size + 1;

        //Defining grid size
        dim3 blocksPerGrid(d1, d2);

        kernel2<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        
        hipDeviceSynchronize();

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
      }
  }