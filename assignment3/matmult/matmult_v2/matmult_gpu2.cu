/*
Implementation using one thread pr. element in C
*/
__global__ void kernel2(int m, int n, int k, double *A, double *B, double *C) {
    double sum = 0.0;
    int i, j, q;

    //Computing the global coordinates of the thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
  
    //Checking for out of bounds
    if (i < m && j < n){
        //Multiplitying a row in A with a column in B, corresponding to one element in C
        for (q = 0; q < k; q++) {
            sum += A[i*k + q] * B[q*n + j];
        }
        C[i*n + j] = sum;
    }
}
  
  
extern "C" {
    
#include <hip/hip_runtime.h>
#include <omp.h>
    #include <stdio.h>
    void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;
        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));
        
        //Copy the input parameters unto the device memory
        double start = omp_get_wtime();
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);
        double seconds = omp_get_wtime() - start;

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        int d1,d2;
        int block_size = 16;
                
        //Assigning a 2D thread grid in each block
        dim3 threadsPerBlock(block_size, block_size);

        //Defining the grid layout depending on the input dimensions
        d1 = (int) (m - 1) / block_size + 1;
        d2 = (int) (n - 1) / block_size + 1;

        //Defining grid size
        dim3 blocksPerGrid(d1, d2);
        // double start = omp_get_wtime();
        kernel2<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);

        hipDeviceSynchronize();
        // double seconds = omp_get_wtime() - start;

        //Copying result to host
        double start2 = omp_get_wtime();
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);
        double seconds2 = omp_get_wtime() - start2;

        seconds = seconds + seconds2;
        // printf("Bandwidth (GB/s): %f", (m*k + n*k)*sizeof(double)/(seconds*1e11));
        printf("Time %f ", seconds);
        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
      }
  }