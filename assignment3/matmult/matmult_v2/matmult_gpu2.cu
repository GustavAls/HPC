/*
Implementation using one thread pr. element in C
*/

#include <hip/hip_runtime.h>
#include <math.h>

__global__ void kernel2(int m, int n, int k, double *A, double *B, double *C) {
    double sum = 0;
    int i, j, q;

    //Computing the global coordinates of the thread
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
  
    //Checking for out of bounds
    if (i < m && j < n){
        //Multiplitying a row in A with a column in B, corresponding to one element in C
        for (q = 0; q < k; q++) {
            sum += A[i*k + q] * B[q*n + j];
        }
        C[i*n + j] = sum;
    }
}
  
  
extern "C" {
    void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;
        int d1,d2;
        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));
                
        //Assigning a 2D thread grid in each block
        dim3 threadsPerBlock(16,16,1);

        //Defining the grid layout depending on the input dimensions
        d1 = ceil((double) m / 16.0);
        d2 = ceil((double) n / 16.0);

        dim3 blocksPerGrid(d1, d2);

        kernel2<<<blocksPerGrid,threadsPerBlock>>>(m, n, k, A_d, B_d, C_d);
        
        hipDeviceSynchronize();

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
      }
  }