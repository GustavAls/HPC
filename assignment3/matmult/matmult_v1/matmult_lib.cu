
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

//Identical to week 1 version, however now using single pointer
extern "C" {
	#include <cblas.h>
	void matmult_lib(int m, int n, int k, double *A, double *B, double *C){
		double alpha = 1.0;
        double beta = 0.0;
		cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans,
              		m, n, k, alpha, A, k, B, n, beta, C, n);
    }
}