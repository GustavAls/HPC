
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <cblas.h>

//Identical to week 1 version, however now using single pointer
extern "C" {
	void matmult_lib(int m, int n, int k, double *A, double *B, double *C){
		double alpha = 1.0;
        double beta = 0.0;
		cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans,
              		m, n, k, alpha, A, k, B, n, beta, C, n);
    }
}