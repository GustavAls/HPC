
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

//Identical to week 1 version, however now using single pointer
extern "C" {
	#include <cblas.h>
    #include <omp.h>
    #include <stdio.h>
	void matmult_lib(int m, int	 n, int k, double *A, double *B, double *C){
		double alpha = 1.0;
        double beta = 0.0;
		double start = omp_get_wtime();
		cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans,
              		m, n, k, alpha, A, k, B, n, beta, C, n);
		double seconds = omp_get_wtime() - start;
		printf("Run time (s): %f", seconds);
    }
}