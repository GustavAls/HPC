
#include <hip/hip_runtime.h>
/*
An implementation of a sequential model which uses only a single thread
*/

__global__ void kernel1(int m, int n, int k, double *A, double *B, double *C) {

    for (int i = 0; i < m; i++) {
        for (int h = 0; h < k; h++){
            for (int j = 0; j < n; j++) {
                C[i*n + j] += A[i*k + h] * B[h*n + j];
            }
        }
    }
}


extern "C" {
    void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C) {
        double* A_d, * B_d, * C_d;

        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        //Run the kernel on the input parameters, using a single thread
        kernel1<<<1,1>>>(m, n, k, A_d, B_d, C_d);

        hipDeviceSynchronize();

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
    }
}