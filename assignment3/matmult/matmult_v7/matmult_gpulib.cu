#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "stdio.h"

extern "C" {

    void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C) {
        double *A_d, *B_d, *C_d;
        double alpha = 1.0;
        double beta = 0.0;

        //Cuda allocate memory on device for matrices
        hipMalloc((void**)&A_d, m*k * sizeof(double));
        hipMalloc((void**)&B_d, k*n * sizeof(double));
        hipMalloc((void**)&C_d, m*n * sizeof(double));

        //Copy the input parameters unto the device memory
        hipMemcpy(A_d, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        //Initialize zeros in the output matrix
        hipMemset(C_d, 0, m*n*sizeof(double));

        hipblasHandle_t handle;
        hipblasCreate(&handle);

        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, &A_d[0], k, &B_d[0], n, &beta, &C_d[0], n);

        hipblasDestroy(handle);

        //Copying result to host
        hipMemcpy(C, C_d, m*n*sizeof(double), hipMemcpyDeviceToHost);

        //Freeing memory allocated
        hipFree(A_d); hipFree(B_d); hipFree(C_d);

    }
}