#include "hip/hip_runtime.h"
//
// File 'example3d.cu' illustrates how to use the functions in 
// alloc3d.h, alloc3d_gpu.h, and transfer3d_gpu.h.
//
#include <stdio.h>
#include <omp.h>
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"

void interchange_memory(double ****a, double ****b){
    double*** temp = *a;
    *a = *b;
    *b = temp;
 }

 __global__ void jacobi_d0(double ***u_d0,double ***uo_d0, double***uo_d1,double ***f_d0, int N, int iterations, double factor, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i > 0 && j > 0 && j < N-1 && k > 0 && k < N-1) { 
        if (i == (N / 2 - 1)) {
            u_d0[i][j][k] = factor * (
                uo_d0[i-1][j][k] + uo_d1[0][j][k] +  // At the boundary between the two devices, we need the next value from d1;
                uo_d0[i][j-1][k] + uo_d0[i][j+1][k] +  // this is why we need the unidirectional read from d1, via hipDeviceEnablePeerAccess()
                uo_d0[i][j][k-1] + uo_d0[i][j][k+1] + 
                delta2*f_d0[i][j][k]);
        } else if (i < (N / 2 - 1)) {
            u_d0[i][j][k] = factor * (
                uo_d0[i-1][j][k] + uo_d0[i+1][j][k] + 
                uo_d0[i][j-1][k] + uo_d0[i][j+1][k] + 
                uo_d0[i][j][k-1] + uo_d0[i][j][k+1] + 
                delta2*f_d0[i][j][k]);
        }
    } 
}
 __global__ void jacobi_d1(double ***u_d1,double ***uo_d1, double***uo_d0,double ***f_d1, int N, int iterations, double factor, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < N/2-1 && j > 0 && j < N-1 && k > 0 && k < N-1) { 
        if (i == 0) {
            u_d1[i][j][k] = factor * (
                uo_d0[N/2-1][j][k] + uo_d1[i+1][j][k] +  // At the boundary between the two devices, we need the previous value from d0;
                uo_d1[i][j-1][k] + uo_d1[i][j+1][k] +  // this is why we need the unidirectional read from d0, via hipDeviceEnablePeerAccess()
                uo_d1[i][j][k-1] + uo_d1[i][j][k+1] + 
                delta2*f_d1[i][j][k]);
        } else if (i > 0) {
            u_d1[i][j][k] = factor * (
                uo_d1[i-1][j][k] + uo_d1[i+1][j][k] + 
                uo_d1[i][j-1][k] + uo_d1[i][j+1][k] + 
                uo_d1[i][j][k-1] + uo_d1[i][j][k+1] + 
                delta2*f_d1[i][j][k]);
        }
    } 
}

int
main(int argc, char *argv[])
{
    // This code allocates a 3d array of size N^3 on the host, and two 
    // 3d arrays of half size on devices 0 and 1, respectively. Then
    // the top part of the host array is transferred to device 0 and the 
    // bottom part to device 1.

    // const int N = 200;            // Dimension N x N x N.
    int N = atoi(argv[1]);
    const long nElms = N * N * N; // Number of elements.
    const int start_T = 20;
    const int iterations = 100;

    double 	***u_h = NULL;
    double 	***u_d0 = NULL;
    double 	***u_d1 = NULL;
    double 	***uo_h = NULL;
    double 	***uo_d0 = NULL;
    double 	***uo_d1 = NULL;
    double 	***f_h = NULL;
    double 	***f_d0 = NULL;
    double 	***f_d1 = NULL;

    // Allocate 3d array in host memory.
    if ( (u_h = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (uo_h = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (f_h = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }

    // Allocate 3d array of half size in device 0 memory.
    if ( (u_d0 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (uo_d0 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (f_d0 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }

    // Allocate 3d array of half size in device 1 memory.
    if ( (u_d1 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }
    // Allocate 3d array of half size in device 1 memory.
    if ( (uo_d1 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }
    // Allocate 3d array of half size in device 1 memory.
    if ( (f_d1 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }

    // Transfer top part to device 0.
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1, 0);
    transfer_3d(u_d0, u_h, N / 2, N, N, hipMemcpyHostToDevice);

    // Transfer bottom part to device 1.
    hipSetDevice(1);
    hipDeviceEnablePeerAccess(0, 0);
    transfer_3d(u_d1, u_h + nElms / 2, N / 2, N, N, hipMemcpyHostToDevice);

    // kernel settings
    dim3 blocksize(8, 8, 8); // 8*8*8 < 1024
    dim3 gridsize( ceil((int) N/blocksize.x),ceil((int) N/blocksize.y),ceil((int) N/blocksize.z) );

    // ... compute ...
    double delta = 2.0/((double)N-1.0);
    double delta2 = delta*delta;
    double factor = 1.0 / 6.0;
    double ts = omp_get_wtime();
    for(int n=0; n < iterations; n++){
        interchange_memory(&uo_d0, &u_d0);
        interchange_memory(&uo_d1, &u_d1);

        hipSetDevice(0);
        jacobi_d0<<<gridsize,blocksize>>>(u_d0, uo_d0, uo_d1, f_d0, N, iterations, factor, delta2);
        hipSetDevice(1);
        jacobi_d1<<<gridsize,blocksize>>>(u_d1, uo_d1, uo_d0, f_d1, N, iterations, factor, delta2);

        hipDeviceSynchronize();
        hipSetDevice(0);
        hipDeviceSynchronize();
    }
    double te = omp_get_wtime() - ts;

    // ... transfer back ...
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1, 0);
    transfer_3d(u_h, u_d0, N / 2, N, N, hipMemcpyDeviceToHost);

    hipSetDevice(1);
    hipDeviceEnablePeerAccess(0, 0);
    transfer_3d(u_h + nElms / 2, u_d1, N / 2, N, N, hipMemcpyDeviceToHost);

    // Print times.
    printf("%d %f\n", N, te);

    // Clean up.
    free(u_h);
    free_gpu(u_d0);
    free_gpu(u_d1);

    // printf("Done\n");
}
