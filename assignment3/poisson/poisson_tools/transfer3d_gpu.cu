#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

void
transfer_3d(double ***dst, double ***src, int m, int n, int k, int flag)
{
    long nPtr = m + m * n;
    long nBlk = k * n * m;

    // we only transfer the value block
    checkCudaErrors( hipMemcpy((double *) dst + nPtr,
                                (double *) src + nPtr,
                                nBlk * sizeof(double),
                                (hipMemcpyKind) flag) );
}

void
transfer_3d_to_1d(double *dst, double ***src, int m, int n, int k, int flag)
{
    long nPtr = m + m * n;
    long nBlk = k * n * m;

    // we only transfer the value block
    checkCudaErrors( hipMemcpy((double *) dst,
                                (double *) src + nPtr,
                                nBlk * sizeof(double),
                                (hipMemcpyKind) flag) );
}

void
transfer_3d_from_1d(double ***dst, double *src, int m, int n, int k, int flag)
{
    long nPtr = m + m * n;
    long nBlk = k * n * m;

    // we only transfer the value block
    checkCudaErrors( hipMemcpy((double *) dst + nPtr,
                                (double *) src,
                                nBlk * sizeof(double),
                                (hipMemcpyKind) flag) );
}
