//
// File 'example3d.cu' illustrates how to use the functions in 
// alloc3d.h, alloc3d_gpu.h, and transfer3d_gpu.h.
//
#include <stdio.h>
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"

int
main(int argc, char *argv[])
{
    // This code allocates a 3d array of size N^3 on the host, and two 
    // 3d arrays of half size on devices 0 and 1, respectively. Then
    // the top part of the host array is transferred to device 0 and the 
    // bottom part to device 1.

    const int N = 200;            // Dimension N x N x N.
    const long nElms = N * N * N; // Number of elements.

    double 	***u_h = NULL;
    double 	***u_d0 = NULL;
    double 	***u_d1 = NULL;

    // Allocate 3d array in host memory.
    if ( (u_h = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }

    // Allocate 3d array of half size in device 0 memory.
    if ( (u_d0 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }

    // Allocate 3d array of half size in device 1 memory.
    if ( (u_d1 = d_malloc_3d_gpu(N / 2, N, N)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }

    // Transfer top part to device 0.
    transfer_3d_from_1d(u_d0, u_h[0][0], N / 2, N, N, hipMemcpyHostToDevice);

    // Transfer bottom part to device 1.
    transfer_3d_from_1d(u_d1, u_h[0][0] + nElms / 2, N / 2, N, N, hipMemcpyHostToDevice);

    // ... compute ...

    // ... transfer back ...

    // Clean up.
    free(u_h);
    free_gpu(u_d0);
    free_gpu(u_d1);

    printf("Done\n");
}
