
#include <hip/hip_runtime.h>
__global__ void jacobi(double ***u_old,double ***u,double ***F, int N, int iterations, double factor, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    u[i][j][k] = factor * (
        u_old[i-1][j][k] + u_old[i+1][j][k] + 
        u_old[i][j-1][k] + u_old[i][j+1][k] + 
        u_old[i][j][k-1] + u_old[i][j][k+1] + 
        delta2*F[i][j][k]);
}
