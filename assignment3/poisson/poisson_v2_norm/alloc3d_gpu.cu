#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

__global__ void d_malloc_3d_gpu_kernel1(double *** array3D, int m, int n, int k)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m) {
        array3D[i] = (double **) array3D + m + i * n;
        //printf("k1: %i | %i\n", i, i* n);
    }
}

__global__ void d_malloc_3d_gpu_kernel2(double *** array3D, int m, int n, int k)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m && j < n) {
        array3D[i][j] = (double *) array3D + m + m * n + (i * k * n) + (j * k);
        //printf("k2: %i %i | %i\n", i, j, (i * k * n) + (j * k));
    }
}

double ***
d_malloc_3d_gpu(int m, int n, int k) {

    if (k <= 0 || n <= 0 || m <= 0)
        return NULL;
    
    double ***array3D; 
    checkCudaErrors( hipMalloc((void**)&array3D, 
                                m * sizeof(double **) +
                                m * n * sizeof(double *) +
                                m * n * k * sizeof(double)) );
    if (array3D == NULL) {
        return NULL;
    }

    dim3 block(16, 16);
    dim3 grid((m + 15) / 16, (n + 15) /16);
    d_malloc_3d_gpu_kernel1<<<grid.x, block.x>>>(array3D, m, n, k);
    d_malloc_3d_gpu_kernel2<<<grid, block>>>(array3D, m, n, k);
    checkCudaErrors( hipDeviceSynchronize() );
    return array3D;
}

void
free_gpu(double ***array3D) {
    checkCudaErrors( hipFree(array3D) );
}
