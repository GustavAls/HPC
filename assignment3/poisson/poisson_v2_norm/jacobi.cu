
#include <hip/hip_runtime.h>
__global__ void jacobi(double ***u,double ***u_old, double *norm,double ***F, int N, int iterations, double factor, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    double d;

    if (i > 0 && i < N-1 && j > 0 && j < N-1 && k > 0 && k < N-1) { 
        u[i][j][k] = factor * (
            u_old[i-1][j][k] + u_old[i+1][j][k] + 
            u_old[i][j-1][k] + u_old[i][j+1][k] + 
            u_old[i][j][k-1] + u_old[i][j][k+1] + 
            delta2*F[i][j][k]);
        d = (u[i][j][k] - u_old[i][j][k]) * (u[i][j][k] - u_old[i][j][k]);
        atomicAdd(norm, d);
    }
}
