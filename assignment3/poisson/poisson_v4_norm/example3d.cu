#include "hip/hip_runtime.h"
//
// File 'example3d.cu' illustrates how to use the functions in
// alloc3d.h, alloc3d_gpu.h, and transfer3d_gpu.h.
//

#include <stdio.h>
#include <omp.h>
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "initialize_data.h"
#include "jacobi.h"

__inline__ __device__ double warpReduceSum(double value);
__inline__ __device__ double blockReduceSum(double value);

void interchange_memory(double ****a, double ****b)
{
    double ***temp = *a;
    *a = *b;
    *b = temp;
}

int main(int argc, char *argv[])
{
    // This code allocates a 3d array of size N^3 on the host, and two
    // 3d arrays of half size on devices 0 and 1, respectively. Then
    // the top part of the host array is transferred to device 0 and the
    // bottom part to device 1.

    // const int N = 16;            // Dimension N x N x N.
    int N = atoi(argv[1]);
    // const long nElms = N * N * N; // Number of elements.
    const int start_T = 20;
    const int iterations = 100;
    const double threshold = 0.01;

    double ***u_h = NULL;
    double ***uo_h = NULL;
    double ***f_h = NULL;
    double ***u_d = NULL;
    double ***uo_d = NULL;
    double ***f_d = NULL;
    double *norm_d, *norm_h;

    // Allocate 3d array in host memory.
    if ((u_h = d_malloc_3d(N, N, N)) == NULL)
    {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ((uo_h = d_malloc_3d(N, N, N)) == NULL)
    {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ((f_h = d_malloc_3d(N, N, N)) == NULL)
    {
        perror("array u: allocation failed");
        exit(-1);
    }
    hipHostMalloc((void **)&norm_h, sizeof(double));
    *norm_h = 10000.0;

    hipSetDevice(0);

    // Allocate 3d array of half size in device 0 memory.
    if ((u_d = d_malloc_3d_gpu(N, N, N)) == NULL)
    {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ((uo_d = d_malloc_3d_gpu(N, N, N)) == NULL)
    {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ((f_d = d_malloc_3d_gpu(N, N, N)) == NULL)
    {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    hipMalloc((void **)&norm_d, sizeof(double));

    // CPU initializes vectors.
    initialize_data(N, u_h, uo_h, f_h, start_T);

    // CPU -> GPU transfer.
    transfer_3d(u_d, u_h, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(uo_d, uo_h, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(f_d, f_h, N, N, N, hipMemcpyHostToDevice);
    hipMemcpy(norm_d, norm_h, sizeof(double), hipMemcpyHostToDevice);

    // kernel settings
    dim3 blocksize(32, 1, 1);                                                                                   // 8*8*8 < 1024
    dim3 gridsize(ceil((double)N / blocksize.x), ceil((double)N / blocksize.y), ceil((double)N / blocksize.z)); // cast into double for decimal

    // CPU controlled loop Jacobi
    double delta = 2.0 / ((double)N - 1.0);
    double delta2 = delta * delta;
    double factor = 1.0 / 6.0;
    int n = 0;
    double ts = omp_get_wtime();
    while (n<iterations && * norm_h> threshold)
    {
        *norm_h = 0.0;

        hipMemcpy(norm_d, norm_h, sizeof(double), hipMemcpyHostToDevice);
        interchange_memory(&uo_d, &u_d);
        jacobi<<<gridsize, blocksize>>>(u_d, uo_d, norm_d, f_d, N, iterations, factor, delta2);
        hipMemcpy(norm_h, norm_d, sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize(); // Synchronize globally between each step
        n++;
    }
    double te = omp_get_wtime() - ts;
    // GPU -> CPU transfer.
    transfer_3d(u_h, u_d, N, N, N, hipMemcpyDeviceToHost);
    transfer_3d(uo_h, uo_d, N, N, N, hipMemcpyDeviceToHost);
    transfer_3d(f_h, f_d, N, N, N, hipMemcpyDeviceToHost);

    // Print times.
    printf("%d %f\n", N, te);

    // Clean up.
    free(u_h);
    free(uo_h);
    free(f_h);
    free_gpu(u_d);
    free_gpu(uo_d);
    free_gpu(f_d);

    // printf("Done\n");
}