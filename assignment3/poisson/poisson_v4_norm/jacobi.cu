
#include <hip/hip_runtime.h>
__inline__ __device__ double warpReduceSum(double value)
{
  for (int i = 16; i > 0; i /= 2)  {
     value += __shfl_down_sync(-1, value, i);
      }
   return value;
  }
  
__inline__ __device__ double
blockReduceSum(double value) {
  __shared__ double smem[32];
  int indexThread = threadIdx.x + threadIdx.y * blockDim.x+ threadIdx.z * blockDim.y* blockDim.x;


  if (indexThread < warpSize) {
    smem[indexThread]=0;
  }
  __syncthreads();

  value =  warpReduceSum(value);

  if   (indexThread % warpSize == 0)
    {
    smem[indexThread / warpSize]=value;
    }
  __syncthreads();
  if (indexThread < warpSize) {
    value=smem[indexThread];
  }
return warpReduceSum(value);}

__global__ void jacobi(double ***u,double ***u_old, double *norm,double ***F, int N, int iterations, double factor, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    double d = 0;

    if (i > 0 && i < N-1 && j > 0 && j < N-1 && k > 0 && k < N-1) { 
        u[i][j][k] = factor * (
            u_old[i-1][j][k] + u_old[i+1][j][k] + 
            u_old[i][j-1][k] + u_old[i][j+1][k] + 
            u_old[i][j][k-1] + u_old[i][j][k+1] + 
            delta2*F[i][j][k]);
        d = (u[i][j][k] - u_old[i][j][k]) * (u[i][j][k] - u_old[i][j][k]);
    }
    blockReduceSum(d);
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 )
    {
      atomicAdd(norm, d);
    }
}
