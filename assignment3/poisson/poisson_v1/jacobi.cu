
#include <hip/hip_runtime.h>
__global__ void jacobi(double ***u_old,double ***u,double ***F, int N, int iterations, double factor, double delta2){

    int i, j, k;

    for (i = 1; i < N-1; i++) {
        for (j = 1; j < N-1; j++) {
            for (k = 1; k < N-1; k++) {
                u[i][j][k] = factor * (
                    u_old[i-1][j][k] + u_old[i+1][j][k] + 
                    u_old[i][j-1][k] + u_old[i][j+1][k] + 
                    u_old[i][j][k-1] + u_old[i][j][k+1] + 
                    delta2*F[i][j][k]);
            }
        }
    }   
}
