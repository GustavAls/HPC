//
// File 'example3d.cu' illustrates how to use the functions in 
// alloc3d.h, alloc3d_gpu.h, and transfer3d_gpu.h.
//
#include <stdio.h>
#include <omp.h>
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "initialize_data.h"
#include "jacobi.h"

void interchange_memory(double ****a, double ****b){
    double*** temp = *a;
    *a = *b;
    *b = temp;
 }
//  void interchange_memory(double ***a, double ***b){
//     double*** temp = a;
//     a = b;
//     b = temp;
//  }

int
main(int argc, char *argv[])
{
    // This code allocates a 3d array of size N^3 on the host, and two 
    // 3d arrays of half size on devices 0 and 1, respectively. Then
    // the top part of the host array is transferred to device 0 and the 
    // bottom part to device 1.

    // const int N = 16;            // Dimension N x N x N.
    int N = atoi(argv[1]);
    const long nElms = N * N * N; // Number of elements.
    const int start_T = 20;
    const int iterations = 10000;
    

    double 	***u_h = NULL;
    double 	***uo_h = NULL;
    double 	***f_h = NULL;
    double 	***u_d = NULL;
    double 	***uo_d = NULL;
    double 	***f_d = NULL;

    hipSetDevice(0);

    // Allocate 3d array in host memory.
    if ( (u_h = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (uo_h = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (f_h = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }

    // Allocate 3d array of half size in device 0 memory.
    if ( (u_d = d_malloc_3d_gpu(N, N, N)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (uo_d = d_malloc_3d_gpu(N, N, N)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (f_d = d_malloc_3d_gpu(N, N, N)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }

    // CPU initializes vectors.
    initialize_data(N, u_h, uo_h, f_h, start_T);

    // CPU -> GPU transfer.
    transfer_3d(u_d, u_h, N, N, N, hipMemcpyHostToDevice);

    // CPU controlled loop Jacobi
    double delta = 2.0/((double)N-1.0);
    double delta2 = delta*delta;
    double factor = 1.0 / 6.0;
    double ts = omp_get_wtime();
    for(int n=0; n < iterations; n++){
        interchange_memory(&uo_d, &u_d);
        jacobi<<<1, 1>>>(u_d, uo_d, f_d, N, iterations, factor, delta2);
        hipDeviceSynchronize(); // Synchronize globally between each step
    }
    double te = omp_get_wtime() - ts;

    // GPU -> CPU transfer.
    transfer_3d(u_h, u_d, N, N, N, hipMemcpyDeviceToHost);

    // Print times.
    printf("%d %f\n", N, te);

    // Clean up.
    free(u_h);
    free(uo_h);
    free(f_h);
    free_gpu(u_d);
    free_gpu(uo_d);
    free_gpu(f_d);

    // printf("Done\n");
}
