#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include "alloc3d.h"
#include "print.h"
#include "initialize_data.h"

#ifdef _JACOBI
#include "jacobi.h"
#endif

#define N_DEFAULT 100

int
main(int argc, char *argv[]) {

    int 	N = N_DEFAULT;
    int 	iter_max = 1000;
    double	tolerance;
    double	start_T;
    int		output_type = 0;
    char	*output_prefix = "poisson_res";
    char        *output_ext    = "";
    char	output_filename[FILENAME_MAX];
    double 	***u    = NULL;
    double 	***F    = NULL;
    double 	***u_old= NULL;


    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    start_T   = atof(argv[3]);  // start T for all inner grid points
    if (argc == 6) {
	    output_type = atoi(argv[5]);  // ouput type
    }

    // // allocate memory
    // if ( (u = d_malloc_3d(N, N, N)) == NULL ) {
    //     perror("array u: allocation failed");
    //     exit(-1);
    // }
    // if ( (u_old = d_malloc_3d(N, N, N)) == NULL ) {
    //     perror("array u: allocation failed");
    //     exit(-1);
    // }
    // if ( (F = d_malloc_3d(N, N, N)) == NULL ) {
    //     perror("array F: allocation failed");
    //     exit(-1);
    // }

    int N2 = N*N;
    double size = N*N*N*sizeof(double);

    // Gpu
    hipSetDevice(0);
    double *d_dummy;
    hipMalloc((void**)&d_dummy,0);

	double *d_u, *d_u_old, *d_F;

    // Allocate on device 
    hipMalloc((void**)&d_u, size);
    hipMalloc((void**)&d_u_old, size);
    hipMalloc((void**)&d_F, size);

    // Pin in host
    hipHostMalloc((void**)&u, size);
    hipHostMalloc((void**)&u_old, size);
    hipHostMalloc((void**)&F, size);

    initialize_data(N, u, u_old, F, start_T);

    // Transfer CPU -> GPU
    transfer_3d_to_1d( d_F, F, N, N, N, hipMemcpyHostToDevice );
	transfer_3d_to_1d( d_u, u, N, N, N, hipMemcpyHostToDevice );
	transfer_3d_to_1d( d_u_old, u_old, N, N, N, hipMemcpyHostToDevice );
    // hipMemcpy( d_f, f, size, hipMemcpyHostToDevice );
	// hipMemcpy( d_u, u, size, hipMemcpyHostToDevice );
	// hipMemcpy( d_u_old, u_old, size, hipMemcpyHostToDevice );

    double start, elapsed;
    int iter;

    
    #ifdef _JACOBI
    start = omp_get_wtime();
    // iter = jacobi(u_old, u, F, N, iter_max);
    elapsed = omp_get_wtime() - start;
    #endif

    printf("%d %f %d \n", N, elapsed, iter);

    // dump  results if wanted 
    switch(output_type) {
	case 0:
	    // no output at all
	    break;
	case 3:
	    output_ext = ".bin";
	    sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
	    fprintf(stderr, "Write binary dump to %s: ", output_filename);
	    print_binary(output_filename, N, u);
	    break;
	case 4:
	    output_ext = ".vtk";
	    sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
	    fprintf(stderr, "Write VTK file to %s: ", output_filename);
	    print_vtk(output_filename, N, u);
	    break;
	default:
	    fprintf(stderr, "Non-supported output type!\n");
	    break;
    }

    // de-allocate memory
    free(u);
    free(F);
    free(u_old);

    return(0);
}
